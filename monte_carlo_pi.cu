
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

extern "C" __global__ void monte_carlo_pi(int *results, unsigned int seed, int num_points) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= num_points) return;

    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    float x = hiprand_uniform(&state);
    float y = hiprand_uniform(&state);

    results[idx] = (x * x + y * y <= 1.0f) ? 1 : 0;
}
